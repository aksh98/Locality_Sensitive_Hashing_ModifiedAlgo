#include "hip/hip_runtime.h"
/**
 *	Locality senstive Hashing CPU code
 *
 *	@created 
 *	@Arvind and Divyanshu
 *
 *	The MIT License (MIT)
 *
 *	Copyright (c) 2017 Arvind and Divyanshu
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy of this software and 
 *	associated documentation files (the "Software"), to deal in the Software without restriction, 
 *	including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
 *	and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
 *	subject to the following conditions:
 *
 *		The above copyright notice and this permission notice shall be included in all copies or 
 *		substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT 
 *	LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. 
 *	IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
 *	WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE 
 *	SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 */
// #include <sstream>
// #include <random>
#include <iostream>
#include <string.h>
#include <array>
#include <unordered_map>
#include <string>
#include <vector>
#include "lsh.h"
#include <sys/time.h>
#include <math.h>
#include <iterator>
#include <algorithm>
#include <map>
#include <thrust/version.h>
#include <thrust/sort.h>
using namespace std;

long get_usecs (void)
{
   struct timeval t;
   gettimeofday(&t,NULL);
   return t.tv_sec*1000000+t.tv_usec;
}

double cosine_similarity(float *A, float *B, unsigned int Vector_Length)
{
    double dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
 	for(unsigned int i = 0u; i < Vector_Length; ++i) {
        dot += A[i] * B[i] ;
        denom_a += A[i] * A[i] ;
        denom_b += B[i] * B[i] ;
    }
    return dot / (sqrt(denom_a) * sqrt(denom_b)) ;
}

int main()
{
	// string path = "../../data/test";
	int p=25,rows=58307,col=512;
	cout << "Enter the value of P \t";
	cin >> p;
	cout << "Enter the value of col as per dataset \t";
	cin >> col;
	int knn=100;
	cout << "Enter the value of k \t";
	cin >> knn;

	long start = get_usecs();
	float * hyperplane= lsh::hyper_plane(p,rows);
	long end = get_usecs();
	double dur = ((double)(end-start))/1000000;
	printf("Hyperplane with memory transfer Time = %f\n",dur);

	
	long tstart = get_usecs();
	string path = "../data/dataset_1024.txt";
	start = get_usecs();
	
	


	float ** dataset=lsh::read_data(path);
	end = get_usecs();
	dur = ((double)(end-start))/1000000;
	float  *datasets = new float[rows*col];
	float ** trans_data;
	trans_data = new float*[col];
	for(int i = 0; i < col; ++i)
    	trans_data[i] = new float[58307];
	
	for (int h = 0; h < rows; h++){
	    for (int w = 0; w < col; w++){
	        datasets[col * h + w] = dataset[h][w];
	        trans_data[w][h]=dataset[h][w];
	    }

	}
	printf("Dataset Load Time = %f\n",dur);
	//  TODO no rows are hardcorded 
	// float * trans_data = new float[col*rows];
	// for(int i = 0; i < rows; ++i)
	// {
	// 	for(int j = 0; j < col; ++j)
 //        {
 //        	trans_data[rows * j + i]=dataset[i][j];
 //     	}
	// }

	
	hipDeviceSynchronize();
 	start = get_usecs();
	float * hash_matrix = lsh::hash_matrix(datasets,hyperplane,p,col,rows);
	end = get_usecs();
	dur = ((double)(end-start))/1000000;
    printf("Hash matrix  Time = %f\n",dur);
    int no_of_buckets=0;
    start = get_usecs();
    table * hasmerge = lsh::hashtable(hash_matrix,p ,col,rows,&no_of_buckets);
    end = get_usecs();
	dur = ((double)(end-start))/1000000;
	printf("Hash table  Time = %f\n",dur);
	start = get_usecs();
	string path1 = "../data/query1.txt";
    float ** query=lsh::read_data(path1);
    float query_trans[1][rows];
	float  *tquery= new float[rows];
	float  *nquery= new float[rows];
    for (int i = 0; i < rows; i++)
    {
    	for (int j = 0; j < 1; j++)
    	{
    		query_trans[j][i]=query[i][j];
	        tquery[rows * j + i] = query[i][j];
	        nquery[1* i + j] = query[i][j];
    	}
     	 
    }
	char host_query[p];
	for(int i = 0; i <p; i++)
	{
		for(int j = 0; j<1; j++)
		{
			float sum=0;
            for(int k = 0; k <rows; k++)
            {
                sum += hyperplane[col*i+k] * nquery[1*k+j]; // 
                if (sum >=0 ){
                	host_query[i] = '1';
                } 
                else{
                	host_query[i] = '0';
                }
            }
        }
	}
	// std::cout<<host_query<<endl;
	// Doing a K nearest search
	
	int *rank;
	int samples[col];
	float *cosine_rank;
	int cosine_values[col]; 
	int k=0;
	// searching for query key with the hashtable key
	for (int i = 0; i < no_of_buckets; ++i)
	{
		if(strcmp(hasmerge[i].key,host_query)!=0)
		{
			// no match of hash code
			rank=lsh::hamming_distance(hasmerge,host_query,p,no_of_buckets,col);
			
			for(i=0;i<no_of_buckets;i++)
			{
				if(rank[i] < 15)
				{
					for(int j=0;j<hasmerge[i].count;j++)
					{
						// cout <<hasmerge[i].values[j] <<"\t";
						samples[k]=hasmerge[i].values[j];
						k++;
					}
					// cout<<endl;
				}
			}
			// int count=0;
			// cosine_rank=lsh::cosine_distance(hasmerge,host_query,p,no_of_buckets,col);
			for (int j = 0; j < k; j++)
			{
				cosine_values[j]=samples[j];
				// cosine_rank[j]=cosine_similarity(trans_data[samples[j]],query_trans[0],rows);
				// ranks[]=samples[k];
				//cout << samples[k]<<"\t"<<cosine_similarity(trans_data[k],query_trans[0],rows)<<"\n";
			}
			cosine_rank=lsh::cosine_distance(datasets,tquery,samples,rows,col,k);
		    // for (int i = 0; i < 10; ++i)
		    // {
		    // 	 code 
		    // 	printf("%f\n", cosine_rank[i]);
		    // }
			thrust::sort_by_key(cosine_rank, cosine_rank + k, cosine_values,thrust::greater<float>());
			for(int i=0;i<knn;i++)
			{
	   			printf("Simalrity %f Sample column %i\n",cosine_rank[i],cosine_values[i]);	
			}
		}
		else
		{
			// int k=0;
			int temp = knn;
			for(int j=0;j<hasmerge[i].count;j++)
			{
				samples[k]=hasmerge[i].values[j];
				k++;
			}
	  		if(k==temp)
	  		{
	  			for (int j = 0; j < k; j++)
				{
					cosine_values[j]=samples[j];
					cosine_rank[j]=cosine_similarity(trans_data[samples[j]],query_trans[0],rows);
					// ranks[cosine_similarity(trans_data[k],query_trans[0],rows)]=samples[k];
				}

	  		}
	  		else
	  		{
	  			int match=i;
		  		rank=lsh::hamming_distance(hasmerge,host_query,p,no_of_buckets,col);
				// int k=0;
				for(i=0;i<no_of_buckets;i++)
				{
					if(i==match)
						continue;
					else if(rank[i] < 15)
					{
						for(int j=0;j<hasmerge[i].count;j++)
						{
							// cout <<hasmerge[i].values[j] <<"\t";
							samples[k]=hasmerge[i].values[j];
							k++;
						}
						// cout<<endl;
					}
				}
				// cosine_rank=lsh::cosine_distance(hasmerge,host_query,p,no_of_buckets,col);
				for (int j = 0; j < k; j++)
				{
					cosine_values[j]=samples[j];
					cosine_rank[j]=cosine_similarity(trans_data[samples[j]],query_trans[0],rows);
					// ranks[]=samples[k];
					//cout << samples[k]<<"\t"<<cosine_similarity(trans_data[k],query_trans[0],rows)<<"\n";
				}				
	  		}
	  		thrust::sort_by_key(cosine_rank, cosine_rank + k, cosine_values,thrust::greater<float>());
  			for(int i=0;i<knn;i++)
			{
	   			printf("Simalrity %f Sample column %i\n",cosine_rank[i],cosine_values[i]);	
			}
		}
	}
	end = get_usecs();
	dur = ((double)(end-start))/1000000;
	printf("\nQuery Search Time = %f\n",dur);
    long tend = get_usecs();
	double tdur = ((double)(tend-tstart))/1000000;
	printf("Total Program Time = %f\n",tdur);
  	return 0;
}
