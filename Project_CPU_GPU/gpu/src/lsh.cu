#include "hip/hip_runtime.h"
/**
 *	Locality senstive Hashing CPU code
 *  Implementation of routine function of LSH
 *
 *	@created 
 *	@Arvind and Divyanshu
 *
 *	The MIT License (MIT)
 *
 *	Copyright (c) 2017 Arvind and Divyanshu
 *
 *	Permission is hereby granted, free of charge, to any person obtaining a copy of this software and 
 *	associated documentation files (the "Software"), to deal in the Software without restriction, 
 *	including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
 *	and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
 *	subject to the following conditions:
 *
 *		The above copyright notice and this permission notice shall be included in all copies or 
 *		substantial portions of the Software.
 *
 *	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT 
 *	LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. 
 *	IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
 *	WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE 
 *	SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 */

#include <iostream>
#include <vector>
#include <random>
#include <functional>
#include <fstream>
#include <sstream>
#include <string>
#include <unordered_map>
#include <array>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include "lsh.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>
#include <iostream>
#include <cstring>

using namespace std;

#define BLOCK_SIZE 16

float **data;
table a[4096];
long get_usecss (void)
{
   struct timeval t;
   gettimeofday(&t,NULL);
   return t.tv_sec*1000000+t.tv_usec;
}
//utility functions
__device__ char * my_strcpy(char *dest, const char src){
  int i = 0;
  do {
    dest[i] = src;
  }
  while (src!= 0);
  return dest;
}
//utility functions
__device__ char * my_strcat(char *dest, const char src){
  int i = 0;
  while (dest[i] != 0) i++;
  my_strcpy(dest+i, src);
  return dest;
}


// Kernel Function for hyperplane
__global__ void cuda_hyperplane_kernel(float *d_data)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int seed = time_t(NULL);
	hiprandState s;
	hiprand_init(seed, blockIdx.x, 0, &s);
	d_data[i] = hiprand_normal(&s);
}
// Kernel Function for hash_matrix without shared memory concept
__global__ void cuda_hashmatrix_kernel(float *A,float *B,float *C,int d_p, int d_col,int d_rows)
{
	float result = 0;
    int Row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
    int Col = blockIdx.x*BLOCK_SIZE + threadIdx.x;
    
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    for (int k = 0; k < (BLOCK_SIZE + d_rows - 1)/BLOCK_SIZE; k++) {
    	if (k*BLOCK_SIZE + threadIdx.x < d_rows && Row < d_p)
             As[threadIdx.y][threadIdx.x] = A[Row*d_rows + k*BLOCK_SIZE + threadIdx.x];
        else
             As[threadIdx.y][threadIdx.x] = 0.0;

        if (k*BLOCK_SIZE + threadIdx.y < d_rows && Col < d_col)
             Bs[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE + threadIdx.y)*d_col + Col];
        else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads(); 

        for (int n = 0; n < BLOCK_SIZE; ++n) 
            // if ((k*BLOCK_SIZE + n < d_rows && Row < d_p) && (k*BLOCK_SIZE + n < d_rows && Col < d_col))
                // result += A[Row*d_rows + k*BLOCK_SIZE + n] * B[(k*BLOCK_SIZE + n)*d_col + Col];
        	result+=As[threadIdx.y][n] * Bs[n][threadIdx.x];

    }

    if (Row < d_p && Col < d_col)
    {
    	if (result >=0){
			C[((blockIdx.y * blockDim.y + threadIdx.y)*d_col)+(blockIdx.x*blockDim.x)+threadIdx.x]=1;		
    	}
    	else
    	{
			C[((blockIdx.y * blockDim.y + threadIdx.y)*d_col)+(blockIdx.x*blockDim.x)+threadIdx.x]=0;
    	}
    
    }     	
}
__global__ void cuda_hashmatrix_merge_kernel(char **A ,table * B,int d_p,int d_col,int buckets)
{
	int p = 25;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    // B[0].key = A[0];
    memcpy(B[0].key, A[0], sizeof(char)*p);
    B[0].count += 1;
    B[0].values[B[0].count]=0;
    __syncthreads();
    if(B[i].key == A[i])
    {
    	B[i].values[B[i].count]=i;
    	B[i].count += 1;
    }
    else
    {
    	memcpy(B[i].key, A[0], sizeof(char)*p);
        B[i].values[B[i].count] = i;
        B[i].count += 1;
    }
}
__global__ void cuda_cosine_Kernel (float *d_dataset,float *d_query,int *d_samples ,float *d_cosine_values, int rows, int col) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int index = d_samples[i];
	float dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
	for(unsigned int j = 0u; j < rows; j++) {
        dot += d_dataset[col * j + index] * d_query[1 * j ] ;
        denom_a += d_dataset[col * j + index] * d_dataset[col * j + index] ;
        denom_b += d_query[1 * j ] * d_query[1 * j ] ;
    }
    d_cosine_values[i] = dot / (sqrt(denom_a) * sqrt(denom_b));
    // dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
    //printf("%f\n", d_cosine_values[i]);
}

__global__ void cuda_hamming_Kernel(table * device_table, char * device_query, int * device_hamming_dist, int p) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int count = 0;
    
    for (int j = 0; j<p; j++) {
        if (device_table[i].key[j] != device_query[j]) {
            count++;
        }
    }
    device_hamming_dist[i] = count;
}

/*	Read input data set and return it as matrix*/
float ** lsh::read_data(std::string filename)
{
	string line;
	int row=0;
	int col=0;	
	data = new float*[58307];
	for(int i = 0; i < 58307; ++i)
    	data[i] = new float[4096];
	
	ifstream myfile (filename);
	if (myfile.is_open())
	{
		while ( getline (myfile,line) )
		{
			std::istringstream iss(line);
			int val;
			col=0;
			while ( iss >> val)
			{
			
				data[row][col]=val;
				col++;
			}
	  		row++;
		}
		myfile.close();
	}
	else cout << "Unable to open file"; 
	// cout << row << endl << col;
	return data;
}

/*	Hyperplane matrix generation algorithm and return the hyperplane matrix [TODO number generated by random algorithm is not in range of 0 to 1] */
float * lsh::hyper_plane(int p, int rows)
{
	float *data;
	data = new float[p*rows];
    // define device variables
    float* d_data;
    // allocate the memory for the deivce varaible
    hipMalloc((void **)&d_data,p*rows*sizeof(float));
    // calling the kernel function
    long start = get_usecss();
    cuda_hyperplane_kernel<<<p*rows/1024,1024>>>(d_data);
	long end = get_usecss();
	double dur = ((double)(end-start))/1000000;
	printf("Hyperplance kernel Compute Time = %f\n",dur);
    // copy back the result and send the output to cpu
	hipMemcpy(data,d_data,p*rows*sizeof(float),hipMemcpyDeviceToHost);	
	return data;
}

float * lsh::hash_matrix(float *dataset , float *hyperplane,int p ,int col, int rows)
{
	hipError_t cudaStatus;
	// host varaibles
	float *data;
	data = new float[p*col];
	// device varaibles 
	float *d_dataset,*d_hyperplane,* d_data = 0;
	// allocate storage for the device
	hipMalloc((void**)&d_hyperplane, sizeof(float) * p * rows);
	hipMalloc((void**)&d_dataset, sizeof(float) * rows * col);
	hipMalloc((void**)&d_data, sizeof(float)* p * col);	
	// hipMemset(d_data, 0, sizeof(float)* p * col);
	// copy input to the device
	cudaStatus=hipMemcpy(d_dataset, dataset, sizeof(float) * rows * col, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! %s",hipGetErrorString(hipGetLastError()));
	}
	cudaStatus=hipMemcpy(d_hyperplane, hyperplane, sizeof(float) * p * rows, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! %s",hipGetErrorString(hipGetLastError()));
	}
	// calling the kernel function
    long start = get_usecss();
    const dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
    const dim3 num_blocks((col + block_size.x - 1)/block_size.x, (p + block_size.y - 1)/block_size.y);
    cuda_hashmatrix_kernel<<<num_blocks,block_size>>>(d_hyperplane,d_dataset,d_data, p, col,rows);
    cudaStatus = hipMemcpy(data, d_data ,sizeof(float)*p*col,hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! %s",hipGetErrorString(hipGetLastError()));
	}	
	long end = get_usecss();
	double dur = ((double)(end-start))/1000000;
	printf("Hash matrix kernel Compute Time = %f\n",dur);
    // copy back the result and send the output to cpu
	return data;
}

table * lsh::hashtable(float *dataset,int p ,int col, int rows , int *no_of_buckets)
{
	// hipError_t cudaStatus;
	// hipError_t err = hipSuccess;
	// host varaibles
	// char *data[col];
	// char (*data)[col] = new char[col][25];
	// char ** data = new char*[col];
	char sum[p];
	// for (int i=0;i<col;i++) data[i] = new char[25];
	// data = new char[col];
	
	int buckets=1;
	int flag=0;	
	for (int i = 0; i < col; i++)
	{	
		for (int j = 0; j < p; j++)
		{
			if (dataset[col * j + i]>0) 
				sum[j] = '1';
			else
				sum[j] = '0';
		}
		
		if(i==0)
		{
			strcpy(a[i].key,sum);
			a[i].values[a[i].count]=i;
			a[i].count++;
			// buckets++;
		}
		else
		{
			flag=0;
			int k;
			for(k=0;k<buckets;k++)
			{
				if(strcmp(a[k].key,sum)==0)
				{
					a[k].values[a[k].count]=i;
					a[k].count++;
					flag=1;
				}
			}
			if(flag==0)
			{
				strcpy(a[k].key,sum);
				a[k].values[a[k].count]=i;
				a[k].count++;
				buckets++;
			}
		}
	}
	int total_counts = 0;
 	for (int i = 0; i< buckets; i++) {
 		// printf("COUNT = %d\n", a[i].count);
 		total_counts=total_counts+a[i].count;
 	}
 	*no_of_buckets=buckets;
	// // device varaibles 
	// char **d_dataset;
	// table *d_data;
	// // allocate storage for the device
	// hipMalloc((void**)&d_dataset, sizeof(char)*col*25);
	// err = hipMalloc((void**)&d_data, sizeof(table)*col);
 //    if (err != hipSuccess)
 //    {
 //        fprintf(stderr, "Failed to allocate hash table (error code %s)!\n", hipGetErrorString(err));
 //        exit(EXIT_FAILURE);
 //    }	
	// // copy input to the device
	// cudaStatus=hipMemcpy(d_dataset, data, sizeof(char) * col, hipMemcpyHostToDevice);
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaMemcpy1 failed! %s",hipGetErrorString(hipGetLastError()));
	// }
	// // calling the kernel function TODO implement correctly the kernel function
 //    long start = get_usecss();
 //    // const dim3 block_size(BLOCK_SIZE,BLOCK_SIZE);
 //    // const dim3 num_blocks((col + block_size.x - 1)/block_size.x, (p + block_size.y - 1)/block_size.y);
 //    cuda_hashmatrix_merge_kernel<<<col/1024,1024>>>(d_dataset,d_data, p, col,buckets);
 //    cudaStatus = hipMemcpy(a, d_data,sizeof(table)*col,hipMemcpyDeviceToHost);
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "cudaMemcpy2 failed! %s",hipGetErrorString(hipGetLastError()));
	// }
	// cout << a[0].key <<"\t" << a[1].key<<"\t" << a[2].key<<"\t"<< a[3].key<<endl;
	// long end = get_usecss();
	// double dur = ((double)(end-start))/1000000;
	// printf("Hash matrix merge kernel Compute Time = %f\n",dur);
 //    // copy back the result and send the output to cpu
	return a;	
}
int * lsh::hamming_distance(table *hash , char query[],int p, int buckets,int col)
{
	// hipError_t err = hipSuccess;
    int * host_hamming_dist;
    host_hamming_dist = (int *) malloc(buckets*sizeof(int));
    //  device variables
    char * device_query;
    table * device_table;
    int * device_hamming_dist;
    // allocating the memory
    hipMalloc((void **) &device_table, col*sizeof(table));
    hipMalloc((void **) &device_query, p*sizeof(char));
    hipMalloc((void **) &device_hamming_dist, buckets*sizeof(int));
    // Copy data to device memory
    hipMemcpy(device_table, hash, col*sizeof(table), hipMemcpyHostToDevice);
    hipMemcpy(device_query, query, p*sizeof(char), hipMemcpyHostToDevice);
    // calling the kernel
    cuda_hamming_Kernel<<<1, buckets>>> (device_table, device_query, device_hamming_dist, p);
    // copy result back
    hipMemcpy(host_hamming_dist, device_hamming_dist, buckets*sizeof(int), hipMemcpyDeviceToHost);

    return host_hamming_dist;
}
float * lsh::cosine_distance(float *dataset,float *query ,int *samples, int rows,int col,int count)
{
	// hipError_t err = hipSuccess;
    float * host_cosine_dist;
    host_cosine_dist = (float *) malloc(col*sizeof(float));
    //  device variables
    float * device_query;
    float * device_data;
    float * device_cosine_dist;
	int *device_samples;
	// printf("%f\n",dataset[0]);
	// printf("%d\n",count);
	// printf("%f\n",query[2]);
	// printf("%d\n",samples[0]);
    // allocating the memory
    hipMalloc((void **) &device_data, col*rows*sizeof(float));
    hipMalloc((void **) &device_query, rows*sizeof(float));
    hipMalloc((void **) &device_samples, col*sizeof(int));
    hipMalloc((void **) &device_cosine_dist, col*sizeof(float));
    // Copy data to device memory
    hipMemcpy(device_data,dataset, col*rows*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_query, query, rows*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_samples, samples, col*sizeof(int), hipMemcpyHostToDevice);
    // calling the kernel
    if (count>1024)
	    cuda_cosine_Kernel<<<1, 1024>>> (device_data,device_query,device_samples , device_cosine_dist, rows,col);
    else	
    	cuda_cosine_Kernel<<<1, count>>> (device_data,device_query,device_samples ,device_cosine_dist, rows,col);
    // copy result back
    hipMemcpy(host_cosine_dist, device_cosine_dist, col*sizeof(float), hipMemcpyDeviceToHost);
    // for (int i = 0; i < 10; ++i)
    // {
    // 	 code 
    // 	printf("%f\n", host_cosine_dist[i]);
    // }
    return host_cosine_dist;
}